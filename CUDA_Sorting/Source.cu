#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <windows.h>
//#include <time.h>
#include <ctime>

void PrintGenInfo();
void FillArray(int data[]);
int* SortCUDA(int* data);
int* SortCPU(int* data);
void SortElements(int* data, int pos, int mod);
void CheckDataOrder(int dataSorted[]);
void PrintError(std::string zone, hipError_t* blob);

//https://stackoverflow.com/questions/5856250/using-nsight-to-debug

#define DATA_AMOUNT 7500
#define BLOCK_AMOUNT 1
#define THREADS_PER_BLOCK 1000 //max 1024
#define THREADS_IN_GRID (BLOCK_AMOUNT*THREADS_PER_BLOCK)	//max 131072 ???

const dim3 blockSize = dim3(BLOCK_AMOUNT, 1, 1);
const dim3 threadsPerBlock = dim3(THREADS_PER_BLOCK, 1, 1);

//DEVICE
__global__
void OddEvenSort(int* data_d, int iterAmount) {
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	
	if ((id *= iterAmount) < DATA_AMOUNT) {
		int mod = 0;
		
		for (int i = 0; i < DATA_AMOUNT; ++i) {
			__syncthreads();
			for (int j = 0; j < iterAmount; j += 2) {
				int pos = id + mod + j;
				if ((pos + 1) < DATA_AMOUNT) {
					int reg1 = data_d[pos];
					int reg2 = data_d[pos + 1];
					if (reg1 > reg2) {
						data_d[pos] = reg2;
						data_d[pos + 1] = reg1;
					}
					/*if (data_d[pos] > data_d[pos + 1]) {
						int temp = data_d[pos];
						data_d[pos] = data_d[pos + 1];
						data_d[pos+1] = temp;
					}*/
				}
				
			}
			if (mod == 0) mod = 1;
			else mod = 0;
		}
		
	}
	
}

__global__
void Nothing(int* data_d, int iterAmount) {
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	id *= iterAmount;

	for (int i = 0; i < iterAmount; i++)
		data_d[id + i] = data_d[id + i + 1] + id + i;
}

//HOST
int main(int argc, char* argv[]) {
	int* data = new int[DATA_AMOUNT];
	int* dataSortedByCUDA = new int[DATA_AMOUNT];
	int* dataSortedByCPU = new int[DATA_AMOUNT];

	PrintGenInfo();
	FillArray(data);

	dataSortedByCUDA = SortCUDA(data);
	CheckDataOrder(dataSortedByCUDA);

	std::cout << std::endl;
	dataSortedByCPU = SortCPU(data);	
	CheckDataOrder(dataSortedByCPU);

	std::cout << std::endl << "fin" << std::endl;
	getchar();
	return 0;
}

//PRINT INFORMATION AND WARNINGS
void PrintGenInfo() {
	SYSTEM_INFO siSysInfo;
	GetSystemInfo(&siSysInfo);
	hipError_t blob;

	int deviceCount = 0, setDevice = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) PrintError("Get1", &blob);
	if (hipSetDevice(setDevice) != hipSuccess) PrintError("Set1", &blob);

	std::cout <<
		"DATA_AMOUNT: " << DATA_AMOUNT << std::endl <<
		"BLOCK_AMOUNT: " << BLOCK_AMOUNT << std::endl <<
		"THREADS_PER_BLOCK: " << THREADS_PER_BLOCK << std::endl <<
		std::endl << "Available devices: " << deviceCount << std::endl <<
		"Device ID in use: <" << setDevice << ">" << std::endl;


	if ((THREADS_PER_BLOCK * 2) < DATA_AMOUNT) {
		int iterAmount = 2;
		while (1) {
			if (THREADS_PER_BLOCK*iterAmount >= DATA_AMOUNT)
				break;
			iterAmount += 2;
		}
		std::cout << std::endl << "WARNING: low thread count forces " << iterAmount/2 << " thread loops" << std::endl;
	}
		
	std::cout << std::endl;
}

//FILL DATA WITH RAND NUMBERS
void FillArray(int data[]) {
	srand(time(NULL));
	for (int i = 0; i < DATA_AMOUNT; i++)
		data[i] = (rand() % DATA_AMOUNT) + 1;

	/*
	std::cout << "Pre sort:   ";
	for (int i = 0; (i < DATA_AMOUNT - 1) && (i < 10); i++)
		std::cout << data[i] << ", ";
	std::cout << data[10] << (DATA_AMOUNT > 10 ? "..." : "") << std::endl << std::endl;
	*/
}

//SORT DATA WITH CUDA USING ODD-EVEN SORTING
int* SortCUDA(int* data) {
	int* dataSorted = new int[DATA_AMOUNT];
	//int* dataSorted = (int*)malloc(DATA_AMOUNT * sizeof(int));
	int* data_d = 0;
	hipError_t blob;
	int iterAmount = 2;
	hipEvent_t start, stop;
	float duration = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	while (1) {
		if (THREADS_PER_BLOCK*iterAmount >= DATA_AMOUNT)
			break;
		iterAmount += 2;
	}

	if ((blob = hipMalloc((void**)&data_d, DATA_AMOUNT * sizeof(int))) != hipSuccess) PrintError("Mal1", &blob);
	if ((blob = hipMemcpy(data_d, data, DATA_AMOUNT * sizeof(int), hipMemcpyHostToDevice)) != hipSuccess) PrintError("Cpy1", &blob);

	hipEventRecord(start);
	OddEvenSort<<<BLOCK_AMOUNT, THREADS_PER_BLOCK>>>(data_d, iterAmount);
	//Nothing<<<BLOCK_AMOUNT, THREADS_PER_BLOCK>>>(data_d, iterAmount);
	hipEventRecord(stop);

	if ((blob = hipMemcpy(dataSorted, data_d, DATA_AMOUNT * sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess) PrintError("Cpy2", &blob);
	if ((blob = hipFree(data_d)) != hipSuccess) PrintError("Free1", &blob);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&duration, start, stop);

	std::cout << "CUDA sorting time: " << duration/1000 << " sec" << std::endl;
	return dataSorted;
}

//SORT DATA WITH CPU USING ODD-EVEN SORTING
int* SortCPU(int* data) {
	std::clock_t start;
	int* dataSorted = data;
	int mod;

	start = std::clock();	//timer start
	for (int i = 0; i < DATA_AMOUNT; ++i) {	//will run DATA_AMOUNT times
		mod = i%2;
		for (int j = 0; j < DATA_AMOUNT-1; j +=2)	//will run DATA_AMOUNT/2 times
			SortElements(dataSorted, j, mod);
	}
	long double duration = (std::clock() - start) / (long double)CLOCKS_PER_SEC;	//timer stop

	std::cout << "CPU sorting time: " << duration << " sec" << std::endl;
	return dataSorted;
}

//SORTING ALGORITHM USED BY CPU
void SortElements(int* data, int pos, int mod) {
	pos += mod;
	if (data[pos] > data[pos+1] && pos + 1 < DATA_AMOUNT) {
		int temp = data[pos];
		data[pos] = data[pos + 1];
		data[pos + 1] = temp;
	}
}

//CHECK RESULT
void CheckDataOrder(int dataSorted[]) {
	/*std::cout << "Post sort:  ";
	for (int i = 0; (i < DATA_AMOUNT - 1) && (i < 10); i++)
		std::cout << dataSorted[i] << ", ";
	std::cout << dataSorted[10] << (DATA_AMOUNT > 10 ? "..." : "") << std::endl;*/

	int unsortedCount = 0;
	bool incorrectValues = false;
	for (int i = 0; i < DATA_AMOUNT-1; i++) {
		if (dataSorted[i] > dataSorted[i + 1]) {
			unsortedCount++;
			//std::cout << "it:" << i << " - ... " << dataSorted[i - 2] << ", " << dataSorted[i - 1] << ", " << dataSorted[i] << ", " << dataSorted[i + 1] << ", " << dataSorted[i + 2] << " ..." << std::endl;
		}
		if (dataSorted[i] > DATA_AMOUNT || dataSorted[i] < 0)
			incorrectValues = true;
	}

	if (unsortedCount > 0)
		std::cout << std::endl << "WARNING: " << unsortedCount << " elements unsorted" << std::endl;

	if (incorrectValues == true)
		std::cout << "WARNING: incorrect array values" << std::endl;
	
	if(!(unsortedCount > 0) && !(dataSorted[0] < 0 || dataSorted[DATA_AMOUNT - 1] < 0))
		std::cout << "list sorted correctly" << std::endl;
}

//CUDA ERROR TYPE PRINT 
void PrintError(std::string zone, hipError_t* blob) {
	std::cout << "ERROR" << std::endl << "   zone: " << zone << std::endl << "   output: " << hipGetErrorString(*blob) << std::endl;
	*blob = hipDeviceReset();
}